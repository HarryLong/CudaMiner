#include "hip/hip_runtime.h"
#include "gpuminer.h"

#include <iostream>
#include <stdio.h>
#include "cudaHeader.h"
#include "../common/timer.h"

#define BLOCKSIZE 16

//__global__ void gpuBinning_01 ( float *mineralData, float *resultingGrid, float stepSize, int gridLength)
//{
//    int globalThreadIdx((blockIdx.x * blockDim.x) + threadIdx.x);

//    int mineralIndex(globalThreadIdx * 3); // Each thread deals with a single mineral (3 floats)

//    float x = mineralData[mineralIndex];
//    float y = mineralData[mineralIndex+1];
//    float value = mineralData[mineralIndex+2];

//    int closestXStop = lroundf(x/stepSize);
//    int closestYStop = lroundf(y/stepSize);

//    float xDifference = fabs(x-(closestXStop*stepSize));
//    float yDifference = fabs(y-(closestYStop*stepSize));

//    bool reachable(sqrt(pow(xDifference,2)+pow(yDifference,2)) < stepSize); // Make sure it is reachable by the drone

//    if(reachable)
//    {
//        resultingGrid[(globalThreadIdx*gridLength*gridLength) + ((gridLength * closestYStop) + closestXStop)] = value;
//    }

//    __syncthreads();
//}

//__global__ void reduceGrid(float* gridData, int gridSize, int nGrids)
//{
//    int tIdx((blockIdx.x * blockDim.x) + threadIdx.x);

//    for(; nGrids > 1 ; nGrids /= 2)
//    {
//        if(tIdx < (nGrids/2 * gridSize))
//        {
//            gridData[tIdx] += gridData[tIdx + ((nGrids/2)*gridSize)];
//        }
//    }
//    __syncthreads();
//}

__global__ void gpuBinning_02 ( float *mineralData, float *resultingGrid, float stepSize, int gridLength, int nMinerals)
{
    int globalThreadIdx((blockIdx.x * blockDim.x) + threadIdx.x);

    int mineralIndex(globalThreadIdx * 3); // Each thread deals with a single mineral (3 floats)

    float x = mineralData[mineralIndex];
    float y = mineralData[mineralIndex+1];
    float value = mineralData[mineralIndex+2];

    int closestXStop = lroundf(x/stepSize);
    int closestYStop = lroundf(y/stepSize);

    float xDifference = fabs(x-(closestXStop*stepSize));
    float yDifference = fabs(y-(closestYStop*stepSize));

    bool reachable(sqrt(pow(xDifference,2)+pow(yDifference,2)) < stepSize/2.f); // Make sure it is reachable by the drone

    if(reachable && globalThreadIdx < nMinerals)
    {
        atomicAdd(&resultingGrid[gridLength * (closestYStop+1) + closestXStop+1], value); // +1 for the padding
    }
}

// TODO: You can drop the accumulatedGrid as data can be overwritten
__global__ void gpuCreateAccumulatedGrid(float* grid, int gridLength, int layer)
{
    int halfPointLayer(gridLength-2); // -2 as there is left-side padding
    int offset(max(0,layer - halfPointLayer)); // Used to deal with the diagonal getting smaller

    int globalThreadIdx((blockIdx.x * blockDim.x) + threadIdx.x);

    if(globalThreadIdx <= layer-(2*offset))
    {
        int x(globalThreadIdx + offset);
        int y(layer-globalThreadIdx-offset);

        int flatIndex(((y+1)*gridLength)+x+1); // +1 for the padding
        int leftIndex(flatIndex-1);
        int upIndex(flatIndex-gridLength);

        grid[flatIndex] += max(grid[leftIndex],grid[upIndex]);
    }
}

GPUMiner::GPUMiner(MiningData * miningData, float stepSize) : Miner(miningData, stepSize), dimblock(BLOCKSIZE*BLOCKSIZE), summary()
{
}

GPUMiner::~GPUMiner()
{
    CUDA_SAFE_CALL( hipFree(dGridData), "Failed to free device grid data");
}

void GPUMiner::printDeviceInfo()
{
    int deviceCount;
    CUDA_SAFE_CALL( hipGetDeviceCount(&deviceCount) , "Failed to get device count cusing hipGetDeviceCount");
    std::cout << "Auto detecting gpu setup..." << std::endl;
    std::cout << "Device count: " << deviceCount << std::endl;

    int currentDevice;
    CUDA_SAFE_CALL( hipGetDevice(&currentDevice), "Failed to get device using hipGetDevice" );

    hipDeviceProp_t deviceProp;
    {
        CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, currentDevice), "Failed to get device properties using hipGetDeviceProperties" );
    }
    std::cout << "Maximum threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
}

void GPUMiner::performBinning()
{
    /**
      TODO:
        * Iteration 1 - All data is copied to global memory [all mineral data + resulting grid]
        * Iteration 2 - Mineral data for a given block is copied into shared memory
        MISC
        * Check whether the vectors can be copied across straight
        * Use memset to zero-ify remaining data on the GPU
     * @brief CUDA_SAFE_CALL
     */
    Timer timer;

    CUDA_SAFE_CALL( hipSetDevice( 0 ), "Unable to set device to use (method hipSetDevice)" );

    // Calculate the required number of blocks and grids
    int nBlocks = ceil(((float)miningData->nMinerals)/((float)(BLOCKSIZE*BLOCKSIZE))); // each thread deals with binning a single mineral
    dim3 dimGrid(nBlocks);

    // MINERAL DATA
    float* dMineralData;
    {
        timer.start();
        CUDA_SAFE_CALL( hipMalloc( &dMineralData, miningData->size ), "Failed to allocate device memory for dMineralData");
        hipDeviceSynchronize();
        timer.stop(summary.mineralAllocation);

        timer.start();
        CUDA_SAFE_CALL( hipMemcpy( dMineralData, miningData->data , miningData->size, hipMemcpyHostToDevice ), "Failed to copy rawMineralData to device");
        hipDeviceSynchronize();
        timer.stop(summary.mineralCopy);
    }

    // GRID DATA
    {
        // Allocate memory
        timer.start();
        CUDA_SAFE_CALL( hipMalloc( &dGridData, grid.size), "Failed to allocate device memory for dGridData");
        hipDeviceSynchronize();
        timer.stop(summary.gridAllocation);

        timer.start();
        CUDA_SAFE_CALL( hipMemset ( &dGridData[grid.length], 0, grid.size-(grid.length*sizeof(float))), "Failed to memset to 0 the mineral data");
        hipDeviceSynchronize();
        timer.stop(summary.gridMemset);

        // Grid Padding
        timer.start();
        CUDA_SAFE_CALL( hipMemset ( dGridData, 0xbf, grid.length*sizeof(float)), "Failed to memset to the grid data");
        for(int i = 1; i < grid.width; i++)
        {
            CUDA_SAFE_CALL( hipMemset ( &dGridData[grid.length*i], 0xbf, sizeof(float)), "Failed to memset to the grid data");
        }
        hipDeviceSynchronize();
        timer.stop(summary.gridPadding);
    }


    // Run the kernel
    {
        timer.start();
        gpuBinning_02<<< dimGrid, dimblock>>>(dMineralData, dGridData, stepSize, grid.length, miningData->nMinerals);
        hipDeviceSynchronize();
        timer.stop(summary.binningKernel);
    }
    // Free the mining data as it is no longer needed
    {
        timer.start();
        CUDA_SAFE_CALL( hipFree(dMineralData), "Failed to free device mineral data");
        hipDeviceSynchronize();
        timer.stop(summary.mineralFree);
    }
    summary.binning = summary.mineralAllocation + summary.mineralCopy + summary.gridAllocation + summary.gridMemset + summary.gridPadding + summary.binningKernel + summary.mineralFree;
}

void GPUMiner::createAccumulatedGrid()
{
    Timer t;

    int nBlocks = ceil(((float)grid.length)/((float)(BLOCKSIZE*BLOCKSIZE))); // The maximum number of threads will be on the diagonal
    dim3 dimGrid(nBlocks);

    {
        t.start();
        for(int i = 0; i <= grid.getPathLength(); i++)
        {
            gpuCreateAccumulatedGrid<<< dimGrid, dimblock >>>(dGridData,grid.length,i);
        }
        hipDeviceSynchronize();
        t.stop(summary.accumulationKernel);
    }

    // Copy the data back from the device
    {
        t.start();
        CUDA_SAFE_CALL( hipMemcpy( grid.data, dGridData , grid.size, hipMemcpyDeviceToHost), "Failed to copy grid data from device to host");
        hipDeviceSynchronize();
        t.stop(summary.gridRetrieval);
    }
    summary.aggregation = summary.accumulationKernel + summary.gridRetrieval;
}

GPUMiningSummary GPUMiner::getRunSummary()
{
    return summary;
}
