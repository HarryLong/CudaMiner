#include "hip/hip_runtime.h"
#include "gpuminer.h"

#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKSIZE 16

#define CUDA_SAFE_CALL( value , errorMsg)                                 \
{                                                                         \
  hipError_t cudaStat = value;                                           \
  if (cudaStat != hipSuccess)                                            \
  {                                                                       \
    std::cerr << "ERROR at line " << __LINE__ << " in file " << __FILE__  \
    << ": " << errorMsg << " | " << hipGetErrorString(cudaStat) <<       \
    std::endl;                                                            \
    exit(EXIT_FAILURE);                                                   \
  }                                                                       \
  cudaCheckError(__LINE__);\
}\

__global__ void gpuBinning ( float *mineralData, float *resultingGrid, float stepSize, int gridLength, int gridWidth)
{
    int globalThreadIdx((blockIdx.x * blockDim.x) + threadIdx.x);

    int mineralIndex(globalThreadIdx * 3); // Each thread deals with a single mineral (3 floats)

    float x = mineralData[mineralIndex];
    float y = mineralData[mineralIndex+1];
    float value = mineralData[mineralIndex+2];

    int closestXStop = lroundf(x/stepSize);
    int closestYStop = lroundf(y/stepSize);

    float xDifference = fabs(x-(closestXStop*stepSize));
    float yDifference = fabs(y-(closestYStop*stepSize));

    bool reachable(sqrt(pow(xDifference,2)+pow(yDifference,2)) < stepSize); // Make sure it is reachable by the drone

    if(reachable)
    {
        resultingGrid[(globalThreadIdx*gridLength*gridWidth) + ((gridLength * closestYStop) + closestXStop)] = value;
    }

    __syncthreads();
}

__global__ void reduceGrid(float* gridData, int gridSize, int nGrids)
{    
    int tIdx((blockIdx.x * blockDim.x) + threadIdx.x);

    for(; nGrids > 1 ; nGrids /= 2)
    {
        if(tIdx < (nGrids/2 * gridSize))
        {
            gridData[tIdx] += gridData[tIdx + ((nGrids/2)*gridSize)];
        }
    }
    __syncthreads();
}

GPUMiner::GPUMiner(MiningData * miningData, float stepSize) : Miner(miningData, stepSize)
{

}

GPUMiner::~GPUMiner()
{

}

void GPUMiner::printDeviceInfo()
{
    int deviceCount;
    CUDA_SAFE_CALL( hipGetDeviceCount(&deviceCount) , "Failed to get device count cusing hipGetDeviceCount");
    std::cout << "Auto detecting gpu setup..." << std::endl;
    std::cout << "Device count: " << deviceCount << std::endl;

    int currentDevice;
    CUDA_SAFE_CALL( hipGetDevice(&currentDevice), "Failed to get device using hipGetDevice" );

    hipDeviceProp_t deviceProp;
    {
        CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, currentDevice), "Failed to get device properties using hipGetDeviceProperties" );
    }
    std::cout << "Maximum threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
}

void GPUMiner::performBinning()
{
    /**
      TODO:
        * Iteration 1 - All data is copied to global memory [all mineral data + resulting grid]
        * Iteration 2 - Mineral data for a given block is copied into shared memory
        MISC
        * Check whether the vectors can be copied across straight
        * Use memset to zero-ify remaining data on the GPU
     * @brief CUDA_SAFE_CALL
     */
    CUDA_SAFE_CALL( hipSetDevice( 0 ), "Unable to set device to use (method hipSetDevice)" );

    // Calculate the required number of blocks and grids
    int nBlocks = ceil(((float)miningData->nMinerals)/((float)(BLOCKSIZE*BLOCKSIZE))); // each thread deals with binning a single mineral
    dim3 dimblock(BLOCKSIZE*BLOCKSIZE);
    dim3 dimGrid(nBlocks);
    int nThreads(nBlocks*BLOCKSIZE*BLOCKSIZE);

    std::cout << "Block dimensions: [" << dimblock.x << "," << dimblock.y << "," << dimblock.z << "]" << std::endl;
    std::cout << "Grid dimensions: [" << dimGrid.x << "," << dimGrid.y << "," << dimGrid.z << "]" << std::endl;

    // Calculate the grid size
    int miningGrid_length(ceil(miningData->baseX/stepSize)+1); // +1 for points x = 0
    int miningGrid_width(ceil(miningData->baseY/stepSize)+1); // +1 for points y = 0
    int miningGrid_noBytes(miningGrid_length*miningGrid_width*sizeof(float));
    int miningGrid_count(nThreads); // One grid for each thread

    // Allocate the memory on the device
    float* dMineralData;
    int mineralData_noBytes(miningData->nMinerals*3*sizeof(float));
    int dMineralData_noBytes(nBlocks*BLOCKSIZE*BLOCKSIZE*3*sizeof(float)); // Fill the blocks
    float* dGridData;

    std::cout << "Single grid size: " << miningGrid_noBytes << " bytes" << std::endl;
    std::cout << "Total grid size: " << miningGrid_noBytes*miningGrid_count << " bytes" << std::endl;

    // Mineral data
    {
        CUDA_SAFE_CALL( hipMalloc( &dMineralData, dMineralData_noBytes ), "Failed to allocate device memory for dMineralData");
        CUDA_SAFE_CALL( hipMemset( (dMineralData), 0, dMineralData_noBytes), "Failed to memset to 0 the mineral data");
    }

    // Grid data
    {
        CUDA_SAFE_CALL( hipMalloc( &dGridData, miningGrid_noBytes*miningGrid_count ), "Failed to allocate device memory for dGridData");
        CUDA_SAFE_CALL( hipMemset ( dGridData, 0, miningGrid_noBytes*miningGrid_count), "Failed to memset to 0 the mineral data");
    }

    // Copy the data across
    {
        CUDA_SAFE_CALL( hipMemcpy( dMineralData, miningData->data , mineralData_noBytes, hipMemcpyHostToDevice ), "Failed to copy rawMineralData to device");
    }

    // Run the kernel
    {
        gpuBinning <<< dimGrid, dimblock >>>(dMineralData, dGridData, stepSize, miningGrid_length, miningGrid_width);
    }

    nBlocks = ceil((miningGrid_length*miningGrid_width*miningGrid_count)/(2*BLOCKSIZE*BLOCKSIZE));
    dimGrid.x = nBlocks;
    std::cout << "Number of blocks for reduction: " << nBlocks << std::endl;

    // Reduce the resulting grid
    {
        reduceGrid<<< dimGrid, dimblock >>>(dGridData, miningGrid_length*miningGrid_width, miningGrid_count);
    }

    // Copy the data from the device
    {
//        float* tmpHMiningData = (float*) malloc(miningGrid_noBytes*miningGrid_count);
//        CUDA_SAFE_CALL(hipMemcpy( tmpHMiningData, dGridData, miningGrid_noBytes*miningGrid_count, hipMemcpyDeviceToHost ), "Failed to copy rawMineralData to device");

//        std::string filename("/home/harry/tmp/allgrids.txt");
//        for(int i = 0; i < nThreads; i++)
//        {
//            memcpy(grid.data, tmpHMiningData+(i*miningGrid_length*miningGrid_width), miningGrid_noBytes);
//            grid.writeToFile(filename);
//        }
        CUDA_SAFE_CALL(hipMemcpy( grid.data, dGridData , miningGrid_noBytes, hipMemcpyDeviceToHost), "Failed to copy grid data back to host!");
    }
}

void GPUMiner::cudaCheckError(int lineNumber)
{
    hipError_t result = hipGetLastError();                                                                                                                                         \
    if ( result != hipSuccess )                                                                                                                                       \
    {
      std::cerr << "CUDA ERROR in file " << __FILE__ << " line " << lineNumber << ": " << result << std::endl;
      exit ( EXIT_FAILURE );
    }
}
