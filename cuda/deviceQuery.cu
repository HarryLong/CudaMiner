#include "hip/hip_runtime.h"
/* NOTE: This file contains sections of code taken from the NVIDIA_CUDA-5.0_Samples
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "deviceQuery.h"

// 32-bit floating-point add, multiply, multiply-add Operations per Clock Cycle per Multiprocessor
// http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions__throughput-native-arithmetic-instructions
SMVal opsF32_A_M_MAD_perMPperCC[] =
{
    { 0x10, 8 },      // Tesla  Generation (SM 1.0) G80 class
    { 0x11, 8 },      // Tesla  Generation (SM 1.1) G8x class
    { 0x12, 8 },      // Tesla  Generation (SM 1.2) G9x class
    { 0x13, 8 },      // Tesla  Generation (SM 1.3) GT200 class
    { 0x20, 32 },     // Fermi  Generation (SM 2.0) GF100 class
    { 0x21, 48 },     // Fermi  Generation (SM 2.1) GF10x class
    { 0x30, 192 },    // Kepler Generation (SM 3.0) GK10x class
    { 0x35, 192 },    // Kepler Generation (SM 3.5) GK11x class
    { -1, -1 }
};

// 64-bit floating-point add, multiply, multiply-add Operations per Clock Cycle per Multiprocessor
// http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#arithmetic-instructions__throughput-native-arithmetic-instructions
SMVal opsF64_A_M_MAD_perMPperCC[] =
{
    { 0x10, 0 },      // Tesla  Generation (SM 1.0) G80 class
    { 0x11, 0 },      // Tesla  Generation (SM 1.1) G8x class
    { 0x12, 0 },      // Tesla  Generation (SM 1.2) G9x class
    { 0x13, 1 },      // Tesla  Generation (SM 1.3) GT200 class
    { 0x20, 16 },     // Fermi  Generation (SM 2.0) GF100 class
    { 0x21, 4 },      // Fermi  Generation (SM 2.1) GF10x class
    { 0x30, 8 },      // Kepler Generation (SM 3.0) GK10x class
    { 0x35, 64 },     // Kepler Generation (SM 3.5) GK11x class
    { -1, -1 }
};


// Defined number of cores for SM of specific compute versions
SMVal coresPerSM[] =
//TODO 8 - replace all 'FIND_REAL_VALUE_IN_REFRENCE' by the correct value source from a reference (try Appendix F of the CUDA C PROGRAMMING GUIDE v5.0)
{
    { 0x10, 8 },      // Tesla Generation  (SM 1.0) G80   class
    { 0x11, 8 },      // Tesla Generation  (SM 1.1) G8x   class
    { 0x12, 8 },      // Tesla Generation  (SM 1.2) G9x   class
    { 0x13, 8 },      // Tesla Generation  (SM 1.3) GT200 class
    { 0x20, 32 },     // Fermi Generation  (SM 2.0) GF100 class
    { 0x21, 48 },     // Fermi Generation  (SM 2.1) GF10x class
    { 0x30, 192 },    // Kepler Generation (SM 3.0) GK10x class
    { 0x35, 192 },    // Kepler Generation (SM 3.5) GK11x class
    { -1, -1 }
};

int getValFromSMVer(int major, int minor, SMVal* vals)
{
  int index = 0;

  while (vals[index].SM != -1)
  {
    int thisSM = ((major << 4) + minor);
    int testSM = vals[index].SM;

    if (vals[index].SM == ((major << 4) + minor))
      return vals[index].value;

    index++;
  }
  
  // If we get here we didn't find the value in the array
  return -1;
}

int printDeviceInfo()
{
  printf( " --== GPGPU workshop CUDA Device Query (Runtime API) ==--\n\n");
  hipError_t return_status;
  int currentDevice;
  int deviceCount;
  char pciBusId[1024];
  hipFuncCache_t cacheConfig;
  float maxGflops;

  // NUMBER OF DEVICES
  {
    CUDA_SAFE_CALL( hipGetDeviceCount(&deviceCount) , "Failed to get device count cusing hipGetDeviceCount");
  }

  if (deviceCount > 0)
  {
    printf("There are %i CUDA enabled devices on this node\n", deviceCount);

    int driverVersion = 0, runtimeVersion = 0;
    // DRIVER VERSION + RUNTIME VERSION
    {
      CUDA_SAFE_CALL( hipDriverGetVersion ( &driverVersion ) , "Failed to get driver version using cudaDriverVersion");
      CUDA_SAFE_CALL( hipRuntimeGetVersion ( &runtimeVersion ) , "Failed to get runtime version using hipRuntimeGetVersion");
    }

    printf("  CUDA Driver Version    %d.%d \n", driverVersion / 1000,
        (driverVersion % 100) / 10);
    printf("  Runtime Version        %d.%d \n", runtimeVersion / 1000,
        (runtimeVersion % 100) / 10);

    for (int device = 0; device < deviceCount; device++)
    {
      hipDeviceProp_t deviceProp;

      {
        CUDA_SAFE_CALL( hipGetDevice(&currentDevice), "Failed to get device using hipGetDevice" );
      }

      // Check if the the current device is 'device'
      if ( currentDevice != device)
      {
        fprintf(stderr, "ERROR: Device not set.\n");
        exit(EXIT_FAILURE);
      }

      {
        CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, device), "Failed to get device properties using hipGetDeviceProperties" );
      }

      {
        CUDA_SAFE_CALL( hipDeviceGetPCIBusId(pciBusId, 1024, device), "Failed to get pci bus id using hipDeviceGetPCIBusId" );
      }

      // Check if pciBusId has been set
      if ( strncmp(pciBusId, "Set me", 6) == 0 )
      {
        int ln = atoi(&(pciBusId[6]));
        fprintf(stderr, "ERROR: The variable 'pciBusId' has not been instantiated. Pleas correct line %d in '%s' by using an API call to get a relevant value.\n", ln, __FILE__); \
        exit(EXIT_FAILURE);
      }

      {
        CUDA_SAFE_CALL( hipDeviceGetCacheConfig(&cacheConfig), "Failed to get pci bus id using hipDeviceGetPCIBusId" );
      }

      printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);
      printf("  CUDA Capability Major.Minor version number:    %d.%d\n",
          deviceProp.major, deviceProp.minor);

      char msg[256];
      sprintf(msg,
          "  Total amount of global memory:                 %.1f GBytes (%llu bytes)\n",
          (float) deviceProp.totalGlobalMem / 1073741824.0f,
          (unsigned long long) deviceProp.totalGlobalMem);
      printf("%s", msg);

      printf(
          "  (%2d) Multiprocessors x (%3d) CUDA Cores/MP:    %d CUDA Cores\n",
          deviceProp.multiProcessorCount,
          getValFromSMVer(deviceProp.major, deviceProp.minor, coresPerSM),
          getValFromSMVer(deviceProp.major, deviceProp.minor, coresPerSM)
              * deviceProp.multiProcessorCount);
      printf(
          "  GPU Clock rate:                                %.0f MHz (%0.2f GHz)\n",
          deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

      // TODO 7 : Calculate Theoretical max GFLOPS's
      {
        // Number of 32-bit Floating-point operations per clock cycle per Multiprocessor
        // 1.x --> 8
        // 2.0 --> 32
        // 2.1 --> 48
        // 3.x --> 192
        int flops_per_clockcycle_per_mp;
        switch(deviceProp.major){
          case 1:
            flops_per_clockcycle_per_mp = 8;
            break;
          case 2:
            switch(deviceProp.minor){
              case 0:
                flops_per_clockcycle_per_mp = 32;
                break;
              case 1:
                flops_per_clockcycle_per_mp = 48;
                break;
            }
            break;
          case 3:
            flops_per_clockcycle_per_mp = 192;
            break;
        }
        
        printf( " floating point operations per clock cycle per multiprocessor: %d\n",
          flops_per_clockcycle_per_mp);
        maxGflops = ((flops_per_clockcycle_per_mp*deviceProp.multiProcessorCount)*(deviceProp.clockRate*1e-6f)); // clockRate in KiloHertz (i.e n thousand clock cycles per second)

        // Well you can easily find this value, try -->  http://www.nvidia.com/object/tesla-servers.html
        // But how is it calculated?
        // This is going to involve some thinking and some research
        // Hint: have a look at Table 2 of the  CUDA C PROGRAMMING GUIDE v5.0 ( http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html )
      }

      // Check if maxGflops has been set
      // if ( maxGflops < 0 )
      // {
      //   fprintf(stderr, "ERROR: The variable 'maxGflops' has not been calculated. Pleas correct line %d in '%s'. Calculate this value don't hard code it!\n", (int)(-1*maxGflops), __FILE__); \
      //   exit(EXIT_FAILURE);
      // }

      printf("  Max Gigaflops :                                %.1f Gflops\n",
           maxGflops);

      // This is supported in CUDA 5.0 (runtime API device properties)
      printf("  Memory Clock rate:                             %.0f Mhz\n",
          deviceProp.memoryClockRate * 1e-3f);
      printf("  Memory Bus Width:                              %d-bit\n",
          deviceProp.memoryBusWidth);

      if (deviceProp.l2CacheSize)
      {
        printf("  L2 Cache Size:                                 %d bytes (%0.1f KBytes)\n",
            deviceProp.l2CacheSize, deviceProp.l2CacheSize / 1024.0f );
      }

      printf(
          "  Max Texture Dimension Size (x,y,z)             1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
          deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
          deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
          deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
      printf(
          "  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, 2D=(%d,%d) x %d\n",
          deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
          deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
          deviceProp.maxTexture2DLayered[2]);

      printf("  Total amount of constant memory:               %lu bytes (%0.1f KBytes)\n",
          deviceProp.totalConstMem, deviceProp.totalConstMem / 1024.0f );
      printf("  Total amount of shared memory per block:       %lu bytes (%0.1f KBytes)\n",
          deviceProp.sharedMemPerBlock, deviceProp.sharedMemPerBlock / 1024.0f );
      printf("  Total number of registers available per block: %d\n",
          deviceProp.regsPerBlock );
      printf("  Warp size:                                     %d\n",
          deviceProp.warpSize);
      printf("  Maximum number of threads per multiprocessor:  %d\n",
          deviceProp.maxThreadsPerMultiProcessor);
      printf("  Maximum number of threads per block:           %d\n",
          deviceProp.maxThreadsPerBlock);
      printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
          deviceProp.maxThreadsDim[2]);
      printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
          deviceProp.maxGridSize[2]);
      printf("  Maximum memory pitch:                          %lu bytes (%0.1f GBytes)\n",
          deviceProp.memPitch, deviceProp.memPitch / 1073741824.0f );
      printf("  Texture alignment:                             %lu bytes\n",
          deviceProp.textureAlignment );
      printf(
          "  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n",
          (deviceProp.deviceOverlap ? "Yes" : "No"),
          deviceProp.asyncEngineCount);
      printf("  Run time limit on kernels:                     %s\n",
          deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
      printf("  Integrated GPU sharing Host Memory:            %s\n",
          deviceProp.integrated ? "Yes" : "No");
      printf("  Support host page-locked memory mapping:       %s\n",
          deviceProp.canMapHostMemory ? "Yes" : "No");
      printf("  Alignment requirement for Surfaces:            %s\n",
          deviceProp.surfaceAlignment ? "Yes" : "No");
      printf("  Device has ECC support:                        %s\n",
          deviceProp.ECCEnabled ? "Enabled" : "Disabled");
#ifdef WIN32
      printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
#endif
      printf("  Device supports Unified Addressing (UVA):      %s\n",
          deviceProp.unifiedAddressing ? "Yes" : "No");
      printf("  PCI Bus:                                       %s\n",
          pciBusId);
      printf("  Device PCI Bus ID / PCI location ID:           %d / %d\n",
          deviceProp.pciBusID, deviceProp.pciDeviceID);

      const char *sComputeMode[] =
          {
              "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
              "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
              "Prohibited (no host thread can use ::hipSetDevice() with this device)",
              "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
              "Unknown", NULL };

      printf("  Device Cache Configuration:                    ");
      if ( cacheConfig == hipFuncCachePreferNone)
        printf("Default function cache configuration, no preference\n");
      else if ( cacheConfig == hipFuncCachePreferShared)
        printf("Prefer larger shared memory and smaller L1 cache\n");
      else if ( cacheConfig == hipFuncCachePreferShared)
        printf("Prefer larger L1 cache and smaller shared memory\n");
      else if ( cacheConfig == hipFuncCachePreferEqual)
        printf("Prefer equal size L1 cache and shared memory\n");

      printf("  Compute Mode:\n");
      printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
    }
  }
  else
  {
    printf("There are no CUDA enabled devices in this node\n");
  }

  return 0;
}

// int main(int argc, char *argv[])
// {
//   printDeviceInfo();
// }
